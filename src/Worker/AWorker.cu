#include "hip/hip_runtime.h"
#include <iostream>

#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "Worker/AWorker.hpp"

#define TIME_FACTOR 1.0f / 60.0f

__device__
float uniformDisToBounds(float input, float min, float max)
{
    return input * (max - min) + min;
}

__global__
void InitRand(hiprandState *d_state)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(0, index, 0, &d_state[index]);
}

void checkCudaError(const char *function)
{
    hipError_t error = hipGetLastError();

    if (error == hipSuccess)
        return;

    const char *name = hipGetErrorName(error);
    const char *string = hipGetErrorString(error);
    std::cout << "In function " << function << "\nError " << name << " : " << string << "\n"; 
}

AWorker::AWorker()
{
    managesBuffer = false;
}

AWorker::AWorker(GLuint VBO, int particleQuantity, int elemSz)
{
    particleQty = particleQuantity;
    threadPerBlocks = THREAD_PER_BLOCK;
    blocks = particleQty / threadPerBlocks + 1;
    elemSize = elemSz;

    hipGraphicsGLRegisterBuffer(&cudaGL_ptr, VBO, hipGraphicsRegisterFlagsNone);
    checkCudaError("Register buffer");

    hipMalloc(&d_state, sizeof(hiprandState) * threadPerBlocks * blocks);
    InitRand<<<blocks, threadPerBlocks>>>(d_state);

    managesBuffer = true;
}

AWorker::AWorker(const AWorker &other)
{
    particleQty = other.particleQty;
    elemSize = other.elemSize;
    threadPerBlocks = other.threadPerBlocks;
    blocks = other.blocks;
    cudaGL_ptr = other.cudaGL_ptr;
    d_state = other.d_state;

    managesBuffer = false;
}

AWorker::AWorker(AWorker &&other)
{
    other.managesBuffer = false;

    particleQty = other.particleQty;
    elemSize = other.elemSize;
    threadPerBlocks = other.threadPerBlocks;
    blocks = other.blocks;
    cudaGL_ptr = other.cudaGL_ptr;
    d_state = other.d_state;

    managesBuffer = true;
}

AWorker::~AWorker()
{
    if (managesBuffer)
    {
        hipGraphicsUnregisterResource(cudaGL_ptr);
        checkCudaError("Unregister resource");
        hipFree(d_state);
    }
}

AWorker &AWorker::operator=(const AWorker &other)
{
    if (this == &other)
        return *this;

    particleQty = other.particleQty;
    elemSize = other.elemSize;
    threadPerBlocks = other.threadPerBlocks;
    blocks = other.blocks;
    cudaGL_ptr = other.cudaGL_ptr;
    d_state = other.d_state;

    managesBuffer = false;

    return *this;
}

AWorker &AWorker::operator=(AWorker &&other)
{
    if (this == &other)
        return *this;

    other.managesBuffer = false;

    particleQty = other.particleQty;
    elemSize = other.elemSize;
    threadPerBlocks = other.threadPerBlocks;
    blocks = other.blocks;
    cudaGL_ptr = other.cudaGL_ptr;
    d_state = other.d_state;

    managesBuffer = true;

    return *this;
}

void AWorker::Map()
{
    size_t bufferSize = particleQty * elemSize * sizeof(float);
    
    hipGraphicsMapResources(1, &cudaGL_ptr);
    checkCudaError("Map resource");
    hipGraphicsResourceGetMappedPointer((void **)&buffer, &bufferSize, cudaGL_ptr);
    checkCudaError("Get Mapped pointer");
}

void AWorker::Unmap()
{
    hipGraphicsUnmapResources(1, &cudaGL_ptr);
    checkCudaError("Unmap resource");
}


#include "hip/hip_runtime.h"
#include <iostream>

#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "Worker/AWorker.hpp"

#define TIME_FACTOR 1.0f / 60.0f

__device__
float uniformDisToBounds(float input, float min, float max)
{
    return input * (max - min) + min;
}

__global__
void InitRand(hiprandState *d_state)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(0, index, 0, &d_state[index]);
}

void checkCudaError(const char *function)
{
    hipError_t error = hipGetLastError();

    if (error == hipSuccess)
        return;

    const char *name = hipGetErrorName(error);
    const char *string = hipGetErrorString(error);
    std::cout << "In function " << function << "\nError " << name << " : " << string << "\n"; 
}

AWorker::AWorker()
{
    managesBuffer = false;
}

AWorker::AWorker(GLuint VBO, int particleQuantity)
{
    particleQty = particleQuantity;

    threadPerBlocks = 1024;
    blocks = particleQty / threadPerBlocks + 1;
    gravityStrength = BASE_GRAVITY;

    hipGraphicsGLRegisterBuffer(&cudaGL_ptr, VBO, hipGraphicsRegisterFlagsNone);
    checkCudaError("Register buffer");

    hipMalloc(&d_state, sizeof(hiprandState) * threadPerBlocks * blocks);
    InitRand<<<blocks, threadPerBlocks>>>(d_state);

    managesBuffer = true;
}

AWorker::AWorker(const AWorker &other)
{
    particleQty = other.particleQty;
    threadPerBlocks = other.threadPerBlocks;
    blocks = other.blocks;
    currentParticle = other.currentParticle;
    particlePerFrame = other.particlePerFrame;
    cudaGL_ptr = other.cudaGL_ptr;
    d_state = other.d_state;
    gravityStrength = other.gravityStrength;

    managesBuffer = false;
}

AWorker::AWorker(AWorker &&other)
{
    other.managesBuffer = false;

    particleQty = other.particleQty;
    threadPerBlocks = other.threadPerBlocks;
    blocks = other.blocks;
    currentParticle = other.currentParticle;
    particlePerFrame = other.particlePerFrame;
    cudaGL_ptr = other.cudaGL_ptr;
    d_state = other.d_state;
    gravityStrength = other.gravityStrength;
        managesBuffer = true;
}

AWorker::~AWorker()
{
    if (managesBuffer)
    {
        hipGraphicsUnregisterResource(cudaGL_ptr);
        checkCudaError("Unregister resource");
        hipFree(d_state);
    }
}

AWorker &AWorker::operator=(const AWorker &other)
{
    if (this == &other)
        return *this;

    particleQty = other.particleQty;
    threadPerBlocks = other.threadPerBlocks;
    blocks = other.blocks;
    currentParticle = other.currentParticle;
    particlePerFrame = other.particlePerFrame;
    cudaGL_ptr = other.cudaGL_ptr;
    d_state = other.d_state;
    gravityStrength = other.gravityStrength;

    managesBuffer = false;

    return *this;
}

AWorker &AWorker::operator=(AWorker &&other)
{
    if (this == &other)
        return *this;

    other.managesBuffer = false;

    particleQty = other.particleQty;
    threadPerBlocks = other.threadPerBlocks;
    blocks = other.blocks;
    currentParticle = other.currentParticle;
    particlePerFrame = other.particlePerFrame;
    cudaGL_ptr = other.cudaGL_ptr;
    d_state = other.d_state;
    gravityStrength = other.gravityStrength;

    managesBuffer = true;

    return *this;
}

void GravityUp()
{
    if (gravityStrength >= MAX_GRAVITY)
        return;
    gracityStrength += 0.1f;
}

void GravityDown()
{
    if (gravityStrength <= MIN_GRAVITY)
        return;
    gracityStrength -= 0.1f;
}

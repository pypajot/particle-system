#include "hip/hip_runtime.h"
#include <iostream>

#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "Worker/AWorker.hpp"

#define TIME_FACTOR 1.0f / 60.0f

__device__
float uniformDisToBounds(float input, float min, float max)
{
    return input * (max - min) + min;
}

__global__
void InitRand(hiprandState *d_state)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(0, index, 0, &d_state[index]);
}

void checkCudaError(const char *function)
{
    hipError_t error = hipGetLastError();

    if (error == hipSuccess)
        return;

    const char *name = hipGetErrorName(error);
    const char *string = hipGetErrorString(error);
    std::cout << "In function " << function << "\nError " << name << " : " << string << "\n"; 
}

AWorker::AWorker()
{
    managesBuffer = false;
}

AWorker::AWorker(GLuint VBO, int particleQuantity)
{
    particleQty = particleQuantity;

    threadPerBlocks = 1024;
    blocks = particleQty / threadPerBlocks + 1;

    hipGraphicsGLRegisterBuffer(&cudaGL_ptr, VBO, hipGraphicsRegisterFlagsNone);
    checkCudaError("Register buffer");

    hipMalloc(&d_state, sizeof(hiprandState) * threadPerBlocks * blocks);
    InitRand<<<blocks, threadPerBlocks>>>(d_state);

    managesBuffer = true;
}

AWorker::AWorker(const AWorker &other)
{
    particleQty = other.particleQty;
    threadPerBlocks = other.threadPerBlocks;
    blocks = other.blocks;
    currentParticle = other.currentParticle;
    particlePerFrame = other.particlePerFrame;
    cudaGL_ptr = other.cudaGL_ptr;
    d_state = other.d_state;

    managesBuffer = false;
}

AWorker::AWorker(AWorker &&other)
{
    other.managesBuffer = false;

    particleQty = other.particleQty;
    threadPerBlocks = other.threadPerBlocks;
    blocks = other.blocks;
    currentParticle = other.currentParticle;
    particlePerFrame = other.particlePerFrame;
    cudaGL_ptr = other.cudaGL_ptr;
    d_state = other.d_state;

    managesBuffer = true;
}

AWorker::~AWorker()
{
    if (managesBuffer)
    {
        hipGraphicsUnregisterResource(cudaGL_ptr);
        checkCudaError("Unregister resource");
        hipFree(d_state);
    }
}

AWorker &AWorker::operator=(const AWorker &other)
{
    if (this == &other)
        return *this;

    particleQty = other.particleQty;
    threadPerBlocks = other.threadPerBlocks;
    blocks = other.blocks;
    currentParticle = other.currentParticle;
    particlePerFrame = other.particlePerFrame;
    cudaGL_ptr = other.cudaGL_ptr;
    d_state = other.d_state;

    managesBuffer = false;

    return *this;
}

AWorker &AWorker::operator=(AWorker &&other)
{
    if (this == &other)
        return *this;

    other.managesBuffer = false;

    particleQty = other.particleQty;
    threadPerBlocks = other.threadPerBlocks;
    blocks = other.blocks;
    currentParticle = other.currentParticle;
    particlePerFrame = other.particlePerFrame;
    cudaGL_ptr = other.cudaGL_ptr;
    d_state = other.d_state;

    managesBuffer = true;

    return *this;
}

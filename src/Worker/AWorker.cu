#include "hip/hip_runtime.h"
#include <iostream>

#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "Worker/AWorker.hpp"

#define TIME_FACTOR 1.0f / 60.0f

void checkCudaError(const char *function)
{
    hipError_t error = hipGetLastError();

    if (error == hipSuccess)
        return;

    const char *name = hipGetErrorName(error);
    const char *string = hipGetErrorString(error);
    std::cerr << "In function " << function << "\nError " << name << " : " << string << "\n"; 
}

__global__
void InitRand(hiprandState *_d_state)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(0, index, 0, &_d_state[index]);
}

__global__ 
void GravityAction(float *buffer, int bufferIndexMax, Gravity *gravity, int stride)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int gravityIndex = blockIdx.y;
    
    if (index >= bufferIndexMax)
        return;

    float *current = buffer + index * stride;

    if (gravity[gravityIndex].active)
    {
        float distanceX = current[0] - gravity[gravityIndex].pos.x;
        float distanceY = current[1] - gravity[gravityIndex].pos.y;
        float distanceZ = current[2] - gravity[gravityIndex].pos.z;
    
        float distance = powf(distanceX, 2) + powf(distanceY, 2) + powf(distanceZ, 2);
    
        float speedFactor = TIME_FACTOR * gravity[gravityIndex].strength / distance;
    
        current[3] -= distanceX * speedFactor;
        current[4] -= distanceY * speedFactor;
        current[5] -= distanceZ * speedFactor;
    }
}

AWorker::AWorker() :
    _particleQty(),
    _elemSize(),
    _threadPerBlocks(),
    _blocks()
{
    _managesBuffer = false;
}

AWorker::AWorker(GLuint VBO, int particleQty, int elemSize) :
    _particleQty(particleQty),
    _threadPerBlocks(THREAD_PER_BLOCK),
    _blocks(_particleQty / _threadPerBlocks + 1),
    _elemSize(elemSize)
{
    hipGraphicsGLRegisterBuffer(&_cudaGL_ptr, VBO, hipGraphicsRegisterFlagsNone);
    checkCudaError("Register buffer");

    hipMalloc(&_d_state, sizeof(hiprandState) * _threadPerBlocks * _blocks);
    InitRand<<<_blocks, _threadPerBlocks>>>(_d_state);

    _managesBuffer = true;
}

AWorker::AWorker(const AWorker &other) :
    _particleQty(other._particleQty),
    _elemSize(other._elemSize),
    _threadPerBlocks(other._threadPerBlocks),
    _blocks(other._blocks)
{
    _cudaGL_ptr = other._cudaGL_ptr;
    _d_state = other._d_state;

    _managesBuffer = false;
}

// AWorker::AWorker(AWorker &&other) :
//     _particleQty(other._particleQty),
//     _elemSize(other._elemSize),
//     _threadPerBlocks(other._threadPerBlocks),
//     _blocks(other._blocks)
// {
//     other._managesBuffer = false;

//     _cudaGL_ptr = other._cudaGL_ptr;
//     _d_state = other._d_state;

//     _managesBuffer = true;
// }

AWorker::~AWorker()
{
    if (_managesBuffer)
    {
        hipGraphicsUnregisterResource(_cudaGL_ptr);
        checkCudaError("Unregister resource");
        hipFree(_d_state);
    }
}

AWorker &AWorker::operator=(const AWorker &other)
{
    if (this == &other)
        return *this;

    _cudaGL_ptr = other._cudaGL_ptr;
    _d_state = other._d_state;

    _managesBuffer = false;

    return *this;
}

// AWorker &AWorker::operator=(AWorker &&other)
// {
//     if (this == &other)
//         return *this;

//     other._managesBuffer = false;

//     _cudaGL_ptr = other._cudaGL_ptr;
//     _d_state = other._d_state;

//     _managesBuffer = true;

//     return *this;
// }

void AWorker::Map()
{
    size_t bufferSize = _particleQty * _elemSize * sizeof(float);
    
    hipGraphicsMapResources(1, &_cudaGL_ptr);
    checkCudaError("Map resource");
    hipGraphicsResourceGetMappedPointer((void **)&_buffer, &bufferSize, _cudaGL_ptr);
    checkCudaError("Get Mapped pointer");
}

void AWorker::Unmap()
{
    hipGraphicsUnmapResources(1, &_cudaGL_ptr);
    checkCudaError("Unmap resource");
}


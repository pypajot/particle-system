#include "hip/hip_runtime.h"
#include <iostream>

#include <cuda_gl_interop.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "Worker/AWorker.hpp"

#define TIME_FACTOR 1.0f / 60.0f

__device__
float uniformDisToBounds(float input, float min, float max)
{
    return input * (max - min) + min;
}

__global__
void InitRand(hiprandState *_d_state)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(0, index, 0, &_d_state[index]);
}

void checkCudaError(const char *function)
{
    hipError_t error = hipGetLastError();

    if (error == hipSuccess)
        return;

    const char *name = hipGetErrorName(error);
    const char *string = hipGetErrorString(error);
    std::cout << "In function " << function << "\nError " << name << " : " << string << "\n"; 
}

AWorker::AWorker()
{
    _managesBuffer = false;
}

AWorker::AWorker(GLuint VBO, int particleQty, int elemSize) :
    _particleQty(particleQty),
    _threadPerBlocks(THREAD_PER_BLOCK),
    _blocks(_particleQty / _threadPerBlocks + 1),
    _elemSize(elemSize)
{
    hipGraphicsGLRegisterBuffer(&_cudaGL_ptr, VBO, hipGraphicsRegisterFlagsNone);
    checkCudaError("Register buffer");

    hipMalloc(&_d_state, sizeof(hiprandState) * _threadPerBlocks * blocks);
    InitRand<<<_blocks, _threadPerBlocks>>>(_d_state);

    _managesBuffer = true;
}

AWorker::AWorker(const AWorker &other) :
    _particleQty(other._particleQty),
    _elemSize(other._elemSize),
    _threadPerBlocks(other._threadPerBlocks),
    _blocks(other._blocks)
{
    _cudaGL_ptr = other._cudaGL_ptr;
    _d_state = other._d_state;

    _managesBuffer = false;
}

AWorker::AWorker(AWorker &&other) :
    _particleQty(other._particleQty),
    _elemSize(other._elemSize),
    _threadPerBlocks(other._threadPerBlocks),
    _blocks(other._blocks)
{
    other._managesBuffer = false;

    _cudaGL_ptr = other._cudaGL_ptr;
    _d_state = other._d_state;

    _managesBuffer = true;
}

AWorker::~AWorker()
{
    if (_managesBuffer)
    {
        hipGraphicsUnregisterResource(_cudaGL_ptr);
        checkCudaError("Unregister resource");
        hipFree(_d_state);
    }
}

AWorker &AWorker::operator=(const AWorker &other)
{
    if (this == &other)
        return *this;

    _cudaGL_ptr = other._cudaGL_ptr;
    _d_state = other._d_state;

    _managesBuffer = false;

    return *this;
}

AWorker &AWorker::operator=(AWorker &&other)
{
    if (this == &other)
        return *this;

    other._managesBuffer = false;

    _cudaGL_ptr = other._cudaGL_ptr;
    _d_state = other._d_state;

    _managesBuffer = true;

    return *this;
}

void AWorker::Map()
{
    size_t bufferSize = _particleQty * _elemSize * sizeof(float);
    
    hipGraphicsMapResources(1, &_cudaGL_ptr);
    checkCudaError("Map resource");
    hipGraphicsResourceGetMappedPointer((void **)&_buffer, &bufferSize, _cudaGL_ptr);
    checkCudaError("Get Mapped pointer");
}

void AWorker::Unmap()
{
    hipGraphicsUnmapResources(1, &_cudaGL_ptr);
    checkCudaError("Unmap resource");
}

